#include "hip/hip_runtime.h"
#include <stdio.h>
#include <getopt.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include "timer.hpp"

#define N 2048

bool print_matrix = false;
bool validation = false;

float a[N][N], b[N][N], c[N][N];
float *d_a, *d_b, *d_c;

__global__ void mat_mul(
    float *A, float *B, float *C,
    int ROW_A, int COL_A, int COL_B)
{
    const int i = blockDim.y * blockIdx.y + threadIdx.y;
    const int j = blockDim.x * blockIdx.x + threadIdx.x;

    float sum = 0.0f;
    if (i < ROW_A && j < COL_B) {
        for (int k = 0; k < COL_A; ++k) {
            sum += A[i * COL_A + k] * B[k * COL_B + j];
        }
        C[i * COL_B + j] = sum;
    }
}

void mat_mul() {
    hipMalloc(&d_a, sizeof(float) * N * N);
    hipMalloc(&d_b, sizeof(float) * N * N);
    hipMalloc(&d_c, sizeof(float) * N * N);

    hipMemcpy(d_a, a, sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N * N, hipMemcpyHostToDevice);

    dim3 num_of_blocks(N / 32, N / 32);
    dim3 threads_per_block(32, 32);
    mat_mul<<<num_of_blocks, threads_per_block>>>(d_a, d_b, d_c, N, N, N);

    hipMemcpy(c, d_c, sizeof(float) * N * N, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

/*
 * ==================================================================
 *                      DO NOT EDIT BELOW THIS LINE
 * ==================================================================
 */

void check_mat_mul() {
    printf("Validating...\n");

    bool is_valid = true;
    float eps = 1e-3;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float s = 0;
            for (int k = 0; k < N; ++k) {
                s += a[i][k] * b[k][j];
            }
            if (fabsf(c[i][j] - s) > eps && (s == 0 || fabsf((c[i][j] - s) / s) > eps)) {
                printf("c[%d][%d] : correct_value = %f, your_value = %f\n", i, j, s, c[i][j]);
                is_valid = false;
            }
        }
    }

    if (is_valid) {
        printf("result: VALID\n");
    } else {
        printf("result: INVALID\n");
    }
}

void generate_mat(float (*a)[N]) {
    for (int i = 0; i < N; ++i) { 
        for (int j = 0; j < N; ++j) {
            a[i][j] = (float)rand() / RAND_MAX - 0.5;
        }
    }
}

void print_mat(float (*a)[N]) {
    for (int i = 0; i < N; ++i) { 
        for (int j = 0; j < N; ++j) {
            printf("%+.3f ", a[i][j]);
        }
        printf("\n");
    }
}

void print_help(const char* prog_name) {
    printf("Usage: %s [-pvh]\n", prog_name);
    printf("OPTIONS\n");
    printf("  -p : print matrix data.\n");
    printf("  -v : validate matrix multiplication.\n");
    printf("  -h : print this page.\n");
}

void parse_opt(int argc, char **argv) {
    int opt;
    while ((opt = getopt(argc, argv, "pvh")) != -1) {
        switch(opt) {
            case 'p':
                print_matrix = true;
                break;
            case 'v':
                validation = true;
                break;
            case 'h':
            default:
                print_help(argv[0]);
                exit(0);
        }
    }
}

int main(int argc, char **argv) {
    parse_opt( argc, argv );

    generate_mat(a);
    generate_mat(b);

    printf("Calculating..."); fflush(stdout);
    timer_start(0);
    mat_mul();
    double elapsed_time = timer_stop(0);
    printf(" done!\n");

    if (print_matrix) {
        printf("MATRIX A:\n"); print_mat(a);
        printf("MATRIX B:\n"); print_mat(b);
        printf("MATRIX C:\n"); print_mat(c);
    }

    if (validation) {
        check_mat_mul();
    } else {
        printf("Validation is skipped.\n");
    }

    printf("Elapsed time: %f sec\n", elapsed_time);

    return 0;
}
