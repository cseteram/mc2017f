
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

double get_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)1e-6 * tv.tv_usec;
}

__global__ void vec_add(int *x, int *y, int *z, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        z[i] = x[i] + y[i];
    }
}

const int n = 1 << 24;
int *h_x, *h_y, *h_z;
int *d_x, *d_y, *d_z;

double write_time, kernel_time, read_time, t;

int main()
{
    /* host buffer setup */
    h_x = (int*)malloc(sizeof(int) * n);
    h_y = (int*)malloc(sizeof(int) * n);
    h_z = (int*)malloc(sizeof(int) * n);
    for (int i = 0; i < n; ++i) {
        h_x[i] = rand() % 100;
        h_y[i] = rand() % 100;
    }

    /* device buffer setup */
    hipMalloc(&d_x, sizeof(int) * n);
    hipMalloc(&d_y, sizeof(int) * n);
    hipMalloc(&d_z, sizeof(int) * n);

    /* host to device memory transfer */
    t = get_time();
    hipMemcpy(d_x, h_x, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, sizeof(int) * n, hipMemcpyHostToDevice);
    write_time = get_time() - t;

    /* kernel execution */
    int threads_per_block = 1024;
    int num_of_blocks = (n + threads_per_block - 1) / threads_per_block;
    t = get_time();
    vec_add<<<num_of_blocks, threads_per_block>>>(d_x, d_y, d_z, n);
    hipDeviceSynchronize();
    kernel_time = get_time() - t;

    /* device to host memory transfer */
    t = get_time();
    hipMemcpy(h_z, d_z, sizeof(int) * n, hipMemcpyDeviceToHost);
    read_time = get_time() - t;

    /* verification */
    for (int i = 0; i < n; ++i) {
        if (h_x[i] + h_y[i] != h_z[i]) {
            printf("Incorrect (i = %d : %d + %d != %d)\n",
                i, h_x[i], h_y[i], h_z[i]);
            break;
        }
    }

    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);

    printf("write buffer: %f seconds\n", write_time);
    printf("kernel: %f seconds\n", kernel_time);
    printf("read buffer: %f seconds\n\n", read_time);

    printf("Finished!\n");
    return 0;
}
